#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cfloat>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std; 

#define HANDLE_ERROR( err )  ( HandleError( err, __FILE__, __LINE__ ) )
#define NUM_THREADS 1024  // maximum number of threads per blocks
#define NUM_TRIALS 100    // number of trials 


void HandleError( hipError_t err, const char *file, int line ) {
  if ( err != hipSuccess ) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
    exit( EXIT_FAILURE );
  }
}


__global__ void shared_memory_sum(float * d_out, const float * d_in)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}



int main( int argc, char* argv[] ) {
  /*
  main program 
  */

  if(argc < 3) {
    cout << "Format: stats_s <size of array> <random seed>" << endl  ;
    cout << "Arguments:" << endl;
    cout << "  size of array - This is the size of the array to be generated and processed\n"  << endl ;
    cout << "  random seed   - This integer will be used to seed the random number\n"  << endl ;        
    cout << "                  generator that will generate the contents of the array\n"  << endl ;     
    cout << "                  to be processed\n"  << endl ;   
    exit(1);
  }

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
  }
  int dev = 0;
  hipSetDevice(dev);

  hipDeviceProp_t devProps;
  if (hipGetDeviceProperties(&devProps, dev) == 0)
  {
      printf("Using device %d:\n", dev);
      printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
             devProps.name, (int)devProps.totalGlobalMem, 
             (int)devProps.major, (int)devProps.minor, 
             (int)devProps.clockRate);
  }

  //seed for randomization
  unsigned int seed = atoi( argv[2] );
  srand( seed );

  // initialize input array on host 
  unsigned int array_size = atoi(argv[1]);
  float sum = 0 ;
  float *h_in= (float*)malloc(array_size*sizeof(float));
  for(int i = 0; i < array_size; i++ ){
    h_in[i] = -1.0f + (float)random()/((float)RAND_MAX/2.0f);
    sum += h_in[i] ;
  }

  // declare device pointers
  float *d_in, *d_intermediate, *d_out;

  // allocate memory on device
  HANDLE_ERROR(hipMalloc((void **) &d_in, array_size*sizeof(float)));   
  HANDLE_ERROR(hipMalloc((void **) &d_intermediate, array_size*sizeof(float)));   
  HANDLE_ERROR(hipMalloc((void **) &d_out, sizeof(float)));   

  // copy array host -> device
  HANDLE_ERROR(hipMemcpy(d_in, h_in, array_size*sizeof(float), hipMemcpyHostToDevice));   

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  // Set number of blocks and number of threads.
  int num_threads = NUM_THREADS  ;
  int num_blocks = array_size/num_threads ;   
  
  // Run kernels 100 times
  for(int i = 0 ; i < NUM_TRIALS; i++){
    // KERNEL CALL 1ST TIME
    shared_memory_sum<<<num_blocks, num_threads, num_threads*sizeof(float)>>>(d_intermediate, d_in);

    // KERNEL CALL 2ND TIME
    num_threads = num_blocks; // launch one thread for each block in prev step
    num_blocks = 1;
    shared_memory_sum<<<num_blocks, num_threads, num_threads*sizeof(float)>>>(d_out, d_intermediate);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float runtime;
  hipEventElapsedTime(&runtime, start, stop);    
  runtime /= NUM_TRIALS; // average runtime of 100 trials


  // copy data from device -> host
  float h_out;
  HANDLE_ERROR(hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost));

  cout << "Statistics for array size:" << array_size << " seed:"<<  seed << endl ; 
  //cout << "sequential sum: " << sum << endl ;
  //cout << "parallel sum:" << h_out << endl ;
  cout << "runtime:" << runtime << " milliseconds" << endl ;


  free(h_in) ;
  hipFree(d_in);
  hipFree(d_intermediate);
  hipFree(d_out);

  return 0;
}
